#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>
#include <random>
#include <vector>

#include "BTRKF78.hpp"
#include "CudaNBodyOde.cuh"
#include "RKEmbeddedCuda.cuh"
#include "RawCudaOutput.cuh"

auto constexpr N = 16;
auto constexpr L = 1.0;
auto constexpr n_var = N * 6;
auto init_state_and_tol() -> std::pair<double*, double*>;

int main() {
  auto t0 = 0.0;
  auto tf = std::sqrt(L * L * L / N);
  std::cout << "End time = " << tf << '\n';
  auto [dev_x0, dev_tol] = init_state_and_tol();
  auto ode = CudaNBodyOde<n_var>{1.0e-3};
  auto output = RawCudaOutputWithProgress<n_var>{};

  cuda_integrate<n_var, BTRKF78, CudaNBodyOde<n_var>,
                 RawCudaOutputWithProgress<n_var>>(dev_x0, t0, tf, dev_tol,
                                                   dev_tol, ode, output);

  auto output_file = std::ofstream{"n_body_output.txt"};
  for (auto i = 0; i < output.times.size(); ++i) {
    output_file << output.times[i];
    for (auto j = 0; j < n_var; ++j) {
      output_file << ',' << output.states[i][j];
    }
    output_file << '\n';
  }

  hipFree(dev_tol);
  hipFree(dev_x0);

  return 0;
}

auto init_state_and_tol() -> std::pair<double*, double*> {
  // Simple Two-Body Orbit
  // auto host_x0 =
  //     std::array{1.0, 0.0, 0.0, -1.0, 0.0, 0.0, 0.0, 0.5, 0.0, 0.0, -0.5,
  //     0.0};
  // Three-Body Figure-8
  // auto host_x0 =
  //    std::array{0.9700436,   -0.24308753, 0.0, -0.9700436,  0.24308753,  0.0,
  //               0.0,         0.0,         0.0, 0.466203685, 0.43236573,  0.0,
  //               0.466203685, 0.43236573,  0.0, -0.93240737, -0.86473146,
  //               0.0};
  // Pythagorean Three-Body
  // auto host_x0 = std::array{1.0, 3.0, 0.0, -2.0, -1.0, 0.0, 1.0, -1.0, 0.0,
  //                           0.0, 0.0, 0.0, 0.0,  0.0,  0.0, 0.0, 0.0,  0.0};
  // Five-Body Double Figure-8
  // auto host_x0 =
  //    std::array{1.657666,  0.0,       0.0, 0.439775,  -0.169717, 0.0,
  //               -1.268608, -0.267651, 0.0, -1.268608, 0.267651,  0.0,
  //               0.439775,  0.169717,  0.0, 0.0,       -0.593786, 0.0,
  //               1.822785,  0.128248,  0.0, 1.271564,  0.168645,  0.0,
  //               -1.271564, 0.168645,  0.0, -1.822785, 0.128248,  0.0};
  // auto const n_var = host_x0.size();
  // 1024-Body Cube
  auto host_x0 = std::vector<double>(n_var);
  auto gen = std::mt19937{0};
  auto dist = std::uniform_real_distribution<double>(0.0, L);
  for (auto i = 0; i < host_x0.size() / 2; ++i) {
    host_x0[i] = dist(gen);
  }
  auto host_tol = std::vector<double>(n_var);
  std::fill(host_tol.begin(), host_tol.end(), 1.0e-10);
  double* dev_x0 = nullptr;
  hipMalloc(&dev_x0, n_var * sizeof(double));
  hipMemcpy(dev_x0, host_x0.data(), n_var * sizeof(double),
             hipMemcpyHostToDevice);
  double* dev_tol = nullptr;
  hipMalloc(&dev_tol, n_var * sizeof(double));
  hipMemcpy(dev_tol, host_tol.data(), n_var * sizeof(double),
             hipMemcpyHostToDevice);
  return {dev_x0, dev_tol};
}
