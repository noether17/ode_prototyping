#include <benchmark/benchmark.h>

#include <numeric>

#include "CudaNBodyOde.cuh"

#define REPEAT2(X) X X
#define REPEAT4(X) REPEAT2(X) REPEAT2(X)
#define REPEAT(X) REPEAT4(X) REPEAT4(X)

auto constexpr n_repetitions = 16;

static void BM_NBodySimple(benchmark::State& state) {
  auto constexpr n_particles = 1024;
  auto constexpr n_var = n_particles * 6;
  auto host_x = []() {
    auto x = std::array<double, n_var>{};
    std::iota(x.begin(), x.end(), 0.0);
    return x;
  }();

  double* dev_x = nullptr;
  hipMalloc(&dev_x, n_var * sizeof(double));
  hipMemcpy(dev_x, host_x.data(), n_var * sizeof(double),
             hipMemcpyHostToDevice);
  double* dev_f = nullptr;
  hipMalloc(&dev_f, n_var * sizeof(double));

  auto simple_n_body = CudaNBodyOdeSimple<n_var>{};
  for (auto _ : state) {
    REPEAT(benchmark::DoNotOptimize(dev_f);
           simple_n_body.compute_rhs(dev_x, dev_f); benchmark::ClobberMemory();)
  }

  hipMemcpy(host_x.data(), dev_f, n_var * sizeof(double),
             hipMemcpyDeviceToHost);
  benchmark::DoNotOptimize(host_x.data());
  benchmark::ClobberMemory();

  state.SetItemsProcessed(state.iterations() * n_particles * n_repetitions);

  hipFree(dev_f);
  hipFree(dev_x);
}

BENCHMARK(BM_NBodySimple);

static void BM_NBodyPairwise(benchmark::State& state) {
  auto constexpr n_particles = 1024;
  auto constexpr n_var = n_particles * 6;
  auto host_x = []() {
    auto x = std::array<double, n_var>{};
    std::iota(x.begin(), x.end(), 0.0);
    return x;
  }();

  double* dev_x = nullptr;
  hipMalloc(&dev_x, n_var * sizeof(double));
  hipMemcpy(dev_x, host_x.data(), n_var * sizeof(double),
             hipMemcpyHostToDevice);
  double* dev_f = nullptr;
  hipMalloc(&dev_f, n_var * sizeof(double));

  auto pairwise_n_body = CudaNBodyOde<n_var>{};
  for (auto _ : state) {
    REPEAT(benchmark::DoNotOptimize(dev_f);
           pairwise_n_body.compute_rhs(dev_x, dev_f);
           benchmark::ClobberMemory();)
  }

  hipMemcpy(host_x.data(), dev_f, n_var * sizeof(double),
             hipMemcpyDeviceToHost);
  benchmark::DoNotOptimize(host_x.data());
  benchmark::ClobberMemory();

  state.SetItemsProcessed(state.iterations() * n_particles * n_repetitions);

  hipFree(dev_f);
  hipFree(dev_x);
}

BENCHMARK(BM_NBodyPairwise);
