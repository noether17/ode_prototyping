#include <gtest/gtest.h>

#include <algorithm>
#include <numeric>
#include <vector>

#include "CudaUtils.cuh"

TEST(CudaUtilsTest, ElementWiseAdd) {
  auto constexpr n = 10;
  auto a = std::vector<double>(n);
  auto b = std::vector<double>(n);
  auto c = std::vector<double>(n);
  std::iota(a.begin(), a.end(), 0.0);
  std::iota(b.begin(), b.end(), 1.0);
  std::transform(a.begin(), a.end(), b.begin(), c.begin(), std::plus<double>());

  auto a_dev = static_cast<double*>(nullptr);
  auto b_dev = static_cast<double*>(nullptr);
  auto c_dev = static_cast<double*>(nullptr);
  hipMalloc(&a_dev, n * sizeof(double));
  hipMalloc(&b_dev, n * sizeof(double));
  hipMalloc(&c_dev, n * sizeof(double));
  hipMemcpy(a_dev, a.data(), n * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(b_dev, b.data(), n * sizeof(double), hipMemcpyHostToDevice);
  elementwise_add<<<1, 1>>>(a_dev, b_dev, c_dev, n);
  auto c_dev_host = std::vector<double>(n);
  hipMemcpy(c_dev_host.data(), c_dev, n * sizeof(double),
             hipMemcpyDeviceToHost);
  hipFree(a_dev);
  hipFree(b_dev);
  hipFree(c_dev);

  ASSERT_EQ(c, c_dev_host);
}
