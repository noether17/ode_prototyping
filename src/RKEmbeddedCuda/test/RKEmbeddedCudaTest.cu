#include "hip/hip_runtime.h"
#include <gtest/gtest.h>

#include <cmath>
#include <memory>
#include <numeric>

#include "RKEmbeddedCuda.cuh"

TEST(RKEmbeddedCudaTest, ComputeErrorTargetTestSmall) {
  auto constexpr n_var = 10;
  auto host_x = std::vector<double>(n_var);
  std::iota(host_x.begin(), host_x.end(), 0.0);
  auto host_rtol = std::vector<double>(n_var);
  std::iota(host_rtol.begin(), host_rtol.end(), 1.0);
  auto host_atol = std::vector<double>(n_var);
  std::iota(host_atol.begin(), host_atol.end(), 2.0);
  double* dev_x = nullptr;
  hipMalloc(&dev_x, n_var * sizeof(double));
  hipMemcpy(dev_x, host_x.data(), n_var * sizeof(double),
             hipMemcpyHostToDevice);
  double* dev_rtol = nullptr;
  hipMalloc(&dev_rtol, n_var * sizeof(double));
  hipMemcpy(dev_rtol, host_rtol.data(), n_var * sizeof(double),
             hipMemcpyHostToDevice);
  double* dev_atol = nullptr;
  hipMalloc(&dev_atol, n_var * sizeof(double));
  hipMemcpy(dev_atol, host_atol.data(), n_var * sizeof(double),
             hipMemcpyHostToDevice);
  double* dev_error_target = nullptr;
  hipMalloc(&dev_error_target, n_var * sizeof(double));

  cuda_compute_error_target<<<num_blocks<n_var>(), block_size>>>(
      dev_x, dev_rtol, dev_atol, dev_error_target, n_var);

  auto host_cuda_result = std::vector<double>(n_var);
  hipMemcpy(host_cuda_result.data(), dev_error_target, n_var * sizeof(double),
             hipMemcpyDeviceToHost);
  for (auto i = 0; i < n_var; ++i) {
    auto host_result = host_atol[i] + host_rtol[i] * std::abs(host_x[i]);
    EXPECT_DOUBLE_EQ(host_result, host_cuda_result[i]);
  }

  hipFree(dev_error_target);
  hipFree(dev_atol);
  hipFree(dev_rtol);
  hipFree(dev_x);
}

TEST(RKEmbeddedCudaTest, ComputeErrorTargetTestLarge) {
  auto constexpr n_var = 1 << 20;
  auto host_x = std::vector<double>(n_var);
  std::iota(host_x.begin(), host_x.end(), 0.0);
  auto host_rtol = std::vector<double>(n_var);
  std::iota(host_rtol.begin(), host_rtol.end(), 1.0);
  auto host_atol = std::vector<double>(n_var);
  std::iota(host_atol.begin(), host_atol.end(), 2.0);
  double* dev_x = nullptr;
  hipMalloc(&dev_x, n_var * sizeof(double));
  hipMemcpy(dev_x, host_x.data(), n_var * sizeof(double),
             hipMemcpyHostToDevice);
  double* dev_rtol = nullptr;
  hipMalloc(&dev_rtol, n_var * sizeof(double));
  hipMemcpy(dev_rtol, host_rtol.data(), n_var * sizeof(double),
             hipMemcpyHostToDevice);
  double* dev_atol = nullptr;
  hipMalloc(&dev_atol, n_var * sizeof(double));
  hipMemcpy(dev_atol, host_atol.data(), n_var * sizeof(double),
             hipMemcpyHostToDevice);
  double* dev_error_target = nullptr;
  hipMalloc(&dev_error_target, n_var * sizeof(double));

  cuda_compute_error_target<<<num_blocks<n_var>(), block_size>>>(
      dev_x, dev_rtol, dev_atol, dev_error_target, n_var);

  auto host_cuda_result = std::vector<double>(n_var);
  hipMemcpy(host_cuda_result.data(), dev_error_target, n_var * sizeof(double),
             hipMemcpyDeviceToHost);
  for (auto i = 0; i < n_var; ++i) {
    auto host_result = host_atol[i] + host_rtol[i] * std::abs(host_x[i]);
    EXPECT_DOUBLE_EQ(host_result, host_cuda_result[i]);
  }

  hipFree(dev_error_target);
  hipFree(dev_atol);
  hipFree(dev_rtol);
  hipFree(dev_x);
}

TEST(RKEmbeddedCudaTest, RKNormTestSmall) {
  auto constexpr n_var = 10;
  auto host_v = std::vector<double>(n_var);
  std::iota(host_v.begin(), host_v.end(), 0.0);
  auto host_scale = std::vector<double>(n_var);
  std::iota(host_scale.begin(), host_scale.end(), 1.0);
  double* dev_v = nullptr;
  hipMalloc(&dev_v, n_var * sizeof(double));
  hipMemcpy(dev_v, host_v.data(), n_var * sizeof(double),
             hipMemcpyHostToDevice);
  double* dev_scale = nullptr;
  hipMalloc(&dev_scale, n_var * sizeof(double));
  hipMemcpy(dev_scale, host_scale.data(), n_var * sizeof(double),
             hipMemcpyHostToDevice);
  double* dev_result = nullptr;
  hipMalloc(&dev_result, sizeof(double));

  cuda_rk_norm<n_var>(dev_v, dev_scale, dev_result);

  auto host_cuda_result = 0.0;
  hipMemcpy(&host_cuda_result, dev_result, sizeof(double),
             hipMemcpyDeviceToHost);
  auto host_result = std::sqrt(
      std::inner_product(host_v.begin(), host_v.end(), host_scale.begin(), 0.0,
                         std::plus<>{},
                         [](auto a, auto b) { return (a / b) * (a / b); }) /
      n_var);
  EXPECT_DOUBLE_EQ(host_result, host_cuda_result);

  hipFree(dev_result);
  hipFree(dev_scale);
  hipFree(dev_v);
}

TEST(RKEmbeddedCudaTest, RKNormTestLarge) {
  auto constexpr n_var = 1 << 20;
  auto host_v = std::vector<double>(n_var);
  std::iota(host_v.begin(), host_v.end(), 0.0);
  auto host_scale = std::vector<double>(n_var);
  std::iota(host_scale.begin(), host_scale.end(), 1.0);
  double* dev_v = nullptr;
  hipMalloc(&dev_v, n_var * sizeof(double));
  hipMemcpy(dev_v, host_v.data(), n_var * sizeof(double),
             hipMemcpyHostToDevice);
  double* dev_scale = nullptr;
  hipMalloc(&dev_scale, n_var * sizeof(double));
  hipMemcpy(dev_scale, host_scale.data(), n_var * sizeof(double),
             hipMemcpyHostToDevice);
  double* dev_result = nullptr;
  hipMalloc(&dev_result, sizeof(double));

  cuda_rk_norm<n_var>(dev_v, dev_scale, dev_result);

  auto host_cuda_result = 0.0;
  hipMemcpy(&host_cuda_result, dev_result, sizeof(double),
             hipMemcpyDeviceToHost);
  auto host_result = std::sqrt(
      std::inner_product(host_v.begin(), host_v.end(), host_scale.begin(), 0.0,
                         std::plus<>{},
                         [](auto a, auto b) { return (a / b) * (a / b); }) /
      n_var);
  EXPECT_DOUBLE_EQ(host_result, host_cuda_result);

  hipFree(dev_result);
  hipFree(dev_scale);
  hipFree(dev_v);
}

TEST(RKEmbeddedCudaTest, EulerStepSmall) {
  auto constexpr n_var = 10;
  auto host_x = std::vector<double>(n_var);
  std::iota(host_x.begin(), host_x.end(), 0.0);
  auto host_f = std::vector<double>(n_var);
  std::iota(host_f.begin(), host_f.end(), 1.0);
  auto host_dt = 0.1;
  double* dev_x0 = nullptr;
  hipMalloc(&dev_x0, n_var * sizeof(double));
  hipMemcpy(dev_x0, host_x.data(), n_var * sizeof(double),
             hipMemcpyHostToDevice);
  double* dev_f0 = nullptr;
  hipMalloc(&dev_f0, n_var * sizeof(double));
  hipMemcpy(dev_f0, host_f.data(), n_var * sizeof(double),
             hipMemcpyHostToDevice);
  double* dev_dt = nullptr;
  hipMalloc(&dev_dt, sizeof(double));
  hipMemcpy(dev_dt, &host_dt, sizeof(double), hipMemcpyHostToDevice);
  double* dev_x1 = nullptr;
  hipMalloc(&dev_x1, n_var * sizeof(double));

  cuda_euler_step<<<num_blocks<n_var>(), block_size>>>(dev_x0, dev_f0, dev_dt,
                                                       dev_x1, n_var);

  auto host_cuda_result = std::vector<double>(n_var);
  hipMemcpy(host_cuda_result.data(), dev_x1, n_var * sizeof(double),
             hipMemcpyDeviceToHost);
  for (auto i = 0; i < n_var; ++i) {
    auto host_result = host_x[i] + host_dt * host_f[i];
    EXPECT_DOUBLE_EQ(host_result, host_cuda_result[i]);
  }

  hipFree(dev_x1);
  hipFree(dev_dt);
  hipFree(dev_f0);
  hipFree(dev_x0);
}

TEST(RKEmbeddedCudaTest, EulerStepLarge) {
  auto constexpr n_var = 1 << 20;
  auto host_x = std::vector<double>(n_var);
  std::iota(host_x.begin(), host_x.end(), 0.0);
  auto host_f = std::vector<double>(n_var);
  std::iota(host_f.begin(), host_f.end(), 1.0);
  auto host_dt = 0.1;
  double* dev_x0 = nullptr;
  hipMalloc(&dev_x0, n_var * sizeof(double));
  hipMemcpy(dev_x0, host_x.data(), n_var * sizeof(double),
             hipMemcpyHostToDevice);
  double* dev_f0 = nullptr;
  hipMalloc(&dev_f0, n_var * sizeof(double));
  hipMemcpy(dev_f0, host_f.data(), n_var * sizeof(double),
             hipMemcpyHostToDevice);
  double* dev_dt = nullptr;
  hipMalloc(&dev_dt, sizeof(double));
  hipMemcpy(dev_dt, &host_dt, sizeof(double), hipMemcpyHostToDevice);
  double* dev_x1 = nullptr;
  hipMalloc(&dev_x1, n_var * sizeof(double));

  cuda_euler_step<<<num_blocks<n_var>(), block_size>>>(dev_x0, dev_f0, dev_dt,
                                                       dev_x1, n_var);

  auto host_cuda_result = std::vector<double>(n_var);
  hipMemcpy(host_cuda_result.data(), dev_x1, n_var * sizeof(double),
             hipMemcpyDeviceToHost);
  for (auto i = 0; i < n_var; ++i) {
    auto host_result = host_x[i] + host_dt * host_f[i];
    EXPECT_DOUBLE_EQ(host_result, host_cuda_result[i]);
  }

  hipFree(dev_x1);
  hipFree(dev_dt);
  hipFree(dev_f0);
  hipFree(dev_x0);
}
