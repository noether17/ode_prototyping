#include "hip/hip_runtime.h"
#include <gtest/gtest.h>

#include <array>
#include <cmath>
#include <memory>
#include <numeric>

#include "RKEmbeddedCuda.cuh"

TEST(RKEmbeddedCudaTest, RKNormTestSmall) {
  auto constexpr n_var = 10;
  auto host_v = std::array<double, n_var>{};
  std::iota(host_v.begin(), host_v.end(), 0.0);
  auto host_scale = std::array<double, n_var>{};
  std::iota(host_scale.begin(), host_scale.end(), 1.0);
  auto dev_v = static_cast<double*>(nullptr);
  hipMalloc(&dev_v, n_var * sizeof(double));
  hipMemcpy(dev_v, host_v.data(), n_var * sizeof(double),
             hipMemcpyHostToDevice);
  auto dev_scale = static_cast<double*>(nullptr);
  hipMalloc(&dev_scale, n_var * sizeof(double));
  hipMemcpy(dev_scale, host_scale.data(), n_var * sizeof(double),
             hipMemcpyHostToDevice);

  auto dev_result = static_cast<double*>(nullptr);
  hipMalloc(&dev_result, sizeof(double));
  auto dev_temp = static_cast<double*>(nullptr);
  hipMalloc(&dev_temp, n_var * sizeof(double));
  rk_norm<n_var>(dev_v, dev_scale, dev_temp, dev_result);
  hipFree(dev_temp);

  auto host_result =
      std::sqrt(std::inner_product(host_v.begin(), host_v.end(),
                                   host_scale.begin(), 0.0, std::plus<>{},
                                   [](auto v, auto scale) {
                                     auto scaled_v = v / scale;
                                     return scaled_v * scaled_v;
                                   }) /
                n_var);
  auto host_cuda_result = 0.0;
  hipMemcpy(&host_cuda_result, dev_result, sizeof(double),
             hipMemcpyDeviceToHost);
  EXPECT_DOUBLE_EQ(host_result, host_cuda_result);
  hipFree(dev_result);
  hipFree(dev_scale);
  hipFree(dev_v);
}

TEST(RKEmbeddedCudaTest, RKNormTestLarge) {
  auto constexpr n_var = 1 << 20;
  auto const tolerance =
      std::numeric_limits<double>::epsilon() * std::log2(n_var);
  auto host_v = std::make_unique<std::array<double, n_var>>();
  std::iota(host_v->begin(), host_v->end(), 0.0);
  auto host_scale = std::make_unique<std::array<double, n_var>>();
  std::iota(host_scale->begin(), host_scale->end(), 1.0);
  auto dev_v = static_cast<double*>(nullptr);
  hipMalloc(&dev_v, n_var * sizeof(double));
  hipMemcpy(dev_v, host_v->data(), n_var * sizeof(double),
             hipMemcpyHostToDevice);
  auto dev_scale = static_cast<double*>(nullptr);
  hipMalloc(&dev_scale, n_var * sizeof(double));
  hipMemcpy(dev_scale, host_scale->data(), n_var * sizeof(double),
             hipMemcpyHostToDevice);

  auto dev_result = static_cast<double*>(nullptr);
  hipMalloc(&dev_result, sizeof(double));
  auto dev_temp = static_cast<double*>(nullptr);
  hipMalloc(&dev_temp, n_var * sizeof(double));
  rk_norm<n_var>(dev_v, dev_scale, dev_temp, dev_result);
  hipFree(dev_temp);

  auto host_result =
      std::sqrt(std::inner_product(host_v->begin(), host_v->end(),
                                   host_scale->begin(), 0.0, std::plus<>{},
                                   [](auto v, auto scale) {
                                     auto scaled_v = v / scale;
                                     return scaled_v * scaled_v;
                                   }) /
                n_var);
  auto host_cuda_result = 0.0;
  hipMemcpy(&host_cuda_result, dev_result, sizeof(double),
             hipMemcpyDeviceToHost);
  EXPECT_NEAR(host_result, host_cuda_result, tolerance);
  hipFree(dev_result);
  hipFree(dev_scale);
  hipFree(dev_v);
}
